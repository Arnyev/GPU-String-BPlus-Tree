#include "hip/hip_runtime.h"
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include ""
#include <stdio.h>
#include "hip/hip_vector_types.h"
#include "thrust/sort.h"
#include <thrust/execution_policy.h>
#include "thrust/device_ptr.h"
