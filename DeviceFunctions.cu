#include "hip/hip_runtime.h"
#include ""
#include "thrust/sort.h"
#include "thrust/device_ptr.h"
#include "parameters.h"
#include "functions.h"
#include <thrust/remove.h>
#include <thrust/unique.h>
#include <thrust/device_malloc.h>
#include <thrust/transform_scan.h>
#include <thrust/device_free.h>
#include "DeviceFunctions.cuh"

using namespace thrust;
using namespace std;

__device__ __host__ __inline__ ullong get_hash(uchar* words, const int chars_to_hash, const int my_position)
{
	uchar last_bit = 1;
	uchar char_mask = CHARMASK;

	ullong hash = 0;

	for (int i = 0; i < chars_to_hash; i++)
	{
		const unsigned char c = words[i + my_position];
		if (c == BREAKCHAR)
		{
			char_mask = 0;
			last_bit = 0;
		}
		hash *= ALPHABETSIZE;
		hash += c & char_mask;
	}
	if (words[chars_to_hash + my_position] == BREAKCHAR)
		last_bit = 0;

	return hash << 1 | last_bit;
}

__global__ void create_hashes_with_seg_d(uchar* words, int* word_positions, int* segments, ullong* keys,
                                         const int offset, const int chars_to_hash, const int seg_shift,
                                         const int word_count)
{
	const int thread_num = threadIdx.x + blockDim.x*blockIdx.x;
	if (thread_num >= word_count)
		return;

	const int position = word_positions[thread_num] + offset;

	keys[thread_num] = static_cast<ullong>(segments[thread_num]) << seg_shift | get_hash(words, chars_to_hash, position);
}

template <class T1>
__global__ void mark_singletons_d(ullong* keys, T1* flags, int* destinations, int* output, int* word_positions,
                                  const int word_count)
{
	const int thread_num = threadIdx.x + blockDim.x*blockIdx.x;
	if (thread_num >= word_count)
		return;

	const ullong key = keys[thread_num];
	const int word_position = word_positions[thread_num];
	const bool finished = (key & 1ULL) == 0ULL;
	const int index_output = destinations[thread_num];

	if (thread_num == 0)
	{
		if (finished || key != keys[thread_num + 1])
		{
			output[index_output] = word_position;
			flags[thread_num] = 0;
		}
		else
			flags[thread_num] = 1;

		return;
	}

	const auto key_last = keys[thread_num - 1];

	if (thread_num == word_count - 1)
	{
		if (key != key_last)
		{
			output[index_output] = word_position;
			flags[thread_num] = 0;
		}
		else if (finished)
		{
			output[index_output] = -1;
			flags[thread_num] = 0;
		}
		else
			flags[thread_num] = 1;

		return;
	}

	const ullong key_next = keys[thread_num + 1];

	if (key != key_last && (finished || key != key_next))
	{
		output[index_output] = word_position;
		flags[thread_num] = 0;
	}
	else if (key == key_last && finished)
	{
		output[index_output] = -1;
		flags[thread_num] = 0;
	}
	else
		flags[thread_num] = 1;
}

__global__ void create_consecutive_numbers_d(int* numbers, const int max_number)
{
	const int thread_num = threadIdx.x + blockDim.x*blockIdx.x;
	if (thread_num >= max_number)
		return;

	numbers[thread_num] = thread_num;
}

template <class T1,class T2>
__global__ void flag_different_than_last_d(T1* keys, T2* segments, const int word_count)
{
	const int thread_num = threadIdx.x + blockDim.x*blockIdx.x;
	if (thread_num >= word_count)
		return;

	segments[thread_num] = thread_num == 0 || keys[thread_num] != keys[thread_num - 1] ? 1 : 0;
}

__global__ void compute_postfix_lengths_d(uchar* words, int* positions, const int word_count, int* lengths)
{
	const int thread_num = threadIdx.x + blockDim.x*blockIdx.x;
	if (thread_num >= word_count)
		return;

	int my_position = positions[thread_num];
	if (my_position == -1)
		return;

	int length = 0;
	uchar c;
	for (int i = 1; i < CHARSTOHASH; i++)
	{
		c = words[my_position + i];
		if (c == BREAKCHAR)
			return;
	}

	my_position = my_position + CHARSTOHASH;
	while(true)
	{
		c = words[my_position];

		if (c == BREAKCHAR)
			break;
		my_position++;
		length++;
	}

	lengths[thread_num] = length + 1;
}

__global__ void copy_suffixes(uchar* words, int* positions, const int word_count, uchar* suffixes,
                              int* suffix_positions)
{
	const int thread_num = threadIdx.x + blockDim.x*blockIdx.x;
	if (thread_num >= word_count)
		return;

	const int suffix_pos = suffix_positions[thread_num];
	const int len = suffix_positions[thread_num + 1] - suffix_pos;
	if (len == 0)
		return;

	const int position = positions[thread_num] + CHARSTOHASH;

	for (int i = 0; i < len; i++)
		suffixes[suffix_pos + i] = words[position + i];
}

template<class T>
struct equal_to_minus_one : thrust::unary_function<T, T>
{
	__host__ __device__ T operator()(const T x) const { return x == -1; }
};

template<class T>
struct equal_to_zero : thrust::unary_function<T, T>
{
	__host__ __device__ T operator()(const T x) const { return x == 0; }
};

struct hash_functor: thrust::unary_function<int, ullong>
{
	uchar* words;

	explicit hash_functor(uchar* words): words(words) {	}

	__host__ __device__ ullong operator()(const int position) const 
	{
		if (position == -1)
			return 0ULL;

		return get_hash(words, CHARSTOHASH, position);
	}
};

struct compute_postfix_length_functor : thrust::unary_function<int, int>
{
	uchar* words;

	__device__  int operator()(int my_position) const
	{
		if (my_position == -1)
			return 0;

		int length = 0;
		uchar c;
		for (int i = 1; i < CHARSTOHASH; i++)
		{
			c = words[my_position + i];
			if (c == BREAKCHAR)
				return 0;
		}

		my_position = my_position + CHARSTOHASH;
		while (true)
		{
			c = words[my_position];

			if (c == BREAKCHAR)
				break;
			my_position++;
			length++;
		}

		return length + 1;
	}
};

void create_hashes(uchar* d_word_array, const device_ptr<int> sorted_positions, const device_ptr<int> positions_end,
                   const device_ptr<ullong> hashes)
{
	hipEvent_t start;
	hipEvent_t stop;
	float milliseconds = 0;

	if (WRITETIME)
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipDeviceSynchronize();
		hipEventRecord(start);
	}

	transform(sorted_positions, positions_end, hashes, hash_functor(d_word_array));

	if (WRITETIME)
	{
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << "Hashes simple took " << milliseconds << " milliseconds" << endl;
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
}

sorting_output create_output(unsigned char* d_word_array, int* d_sorted_positions, int word_count)
{
	const device_ptr<int> sorted_positions(d_sorted_positions);

	const auto positions_end = remove_if(sorted_positions, device_ptr<int>(d_sorted_positions + word_count), equal_to_minus_one<int>());

	word_count = positions_end - sorted_positions;

	uint num_threads, num_blocks;
	compute_grid_size(word_count, BLOCKSIZE, num_blocks, num_threads);

	compute_postfix_length_functor postfix_functor;
	postfix_functor.words = d_word_array;

	const auto suffix_positions = device_malloc<int>(word_count + 1);
	transform_exclusive_scan(sorted_positions, positions_end + 1, suffix_positions, postfix_functor, 0, thrust::plus<int>());

	int output_size;
	checkCudaErrors(hipMemcpy(&output_size, suffix_positions.get() + word_count, sizeof(int), hipMemcpyDeviceToHost));

	auto suffixes = device_malloc<uchar>(output_size);
	copy_suffixes << <num_blocks, num_threads >> > (d_word_array, d_sorted_positions, word_count, suffixes.get(), suffix_positions.get());

	const auto hashes = device_malloc<ullong>(word_count);
	create_hashes(d_word_array, sorted_positions, positions_end, hashes);

	const auto hashes_end = unique_by_key(hashes, hashes + word_count, suffix_positions);
	const int hashes_count = hashes_end.first - hashes;

	return { hashes.get(), suffix_positions.get(), suffixes.get(), hashes_count, output_size };
}

__global__ void reposition_strings_d(unsigned char* d_word_array_in, unsigned char* d_word_array_out,
                                     int* d_position_in, int* d_position_out, const int word_count)
{
	const int thread_num = threadIdx.x + blockDim.x*blockIdx.x;
	if (thread_num >= word_count)
		return;

	const int position_in = d_position_in[thread_num];
	const int position_out = d_position_out[thread_num];

	int i = 0;
	char c;
	do
	{
		c = d_word_array_in[position_in + i];
		d_word_array_out[position_out + i] = c;
		i++;
	} while (c != BREAKCHAR);
}

template <class T1>
void mark_singletons(ullong* d_keys, int* d_word_positions, int* d_destinations, T1* d_flags, int* d_output, int current_count)
{
	hipEvent_t start;
	hipEvent_t stop;
	float milliseconds = 0;

	if (WRITETIME)
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipDeviceSynchronize();
		hipEventRecord(start);
	}

	uint num_threads;
	uint num_blocks;
	compute_grid_size(current_count, BLOCKSIZE, num_blocks, num_threads);
	mark_singletons_d << <num_blocks, num_threads >> > (d_keys, d_flags, d_destinations, d_output, d_word_positions, current_count);
	getLastCudaError("Singletons failed.");

	if (WRITETIME)
	{
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << "Singletons took " << milliseconds << " milliseconds" << endl;
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
}

template <class T>
void flags_different_than_last(ullong* d_keys, T* d_flags, int current_count)
{
	hipEvent_t start;
	hipEvent_t stop;
	float milliseconds = 0;

	if (WRITETIME)
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipDeviceSynchronize();
		hipEventRecord(start);
	}

	uint num_threads;
	uint num_blocks;
	compute_grid_size(current_count, BLOCKSIZE, num_blocks, num_threads);

	flag_different_than_last_d << <num_blocks, num_threads >> > (d_keys, d_flags, current_count);
	getLastCudaError("Finding different flags failed.");

	if (WRITETIME)
	{
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << "Finding different than last took " << milliseconds << " milliseconds" << endl;
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
}

int compute_segment_size(int* d_helper, const int current_count)
{
	int max_segment;
	checkCudaErrors(hipMemcpy(&max_segment, d_helper + current_count - 1, sizeof(int), hipMemcpyDeviceToHost));
	return get_segment_size(max_segment);
}

void create_hashes_with_seg(ullong* d_keys, unsigned char* d_word_array, int* d_word_positions, int* d_helper,
                            const int offset, const int segment_size, const int current_count, const int seg_chars)
{
	hipEvent_t start;
	hipEvent_t stop;
	float milliseconds = 0;

	if (WRITETIME)
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipDeviceSynchronize();
		hipEventRecord(start);
	}

	uint num_threads;
	uint num_blocks;
	compute_grid_size(current_count, BLOCKSIZE, num_blocks, num_threads);
	create_hashes_with_seg_d << <num_blocks, num_threads >> > (d_word_array, d_word_positions, d_helper, d_keys, offset,
	                                                           CHARSTOHASH - seg_chars, KEYBITS - segment_size, current_count);
	getLastCudaError("Creating hashes failed.");

	if (WRITETIME)
	{
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << "Create hashes took " << milliseconds << " milliseconds" << endl;
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
}

void create_consecutive_numbers(const int word_count, int* d_destinations)
{	
	hipEvent_t start;
	hipEvent_t stop;
	float milliseconds = 0;

	if (WRITETIME)
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipDeviceSynchronize();
		hipEventRecord(start);
	}

	uint num_threads;
	uint num_blocks;
	compute_grid_size(word_count, BLOCKSIZE, num_blocks, num_threads);
	create_consecutive_numbers_d << <num_blocks, num_threads >> > (d_destinations, word_count);
	getLastCudaError("Consecutive numbers failed.");

	if (WRITETIME)
	{
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << "Consecutive numbers took " << milliseconds << " milliseconds" << endl;
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
}

void sort_wrapper(const device_ptr<ullong> keys, const device_ptr<int> positions, const int current_count)
{
	hipEvent_t start;
	hipEvent_t stop;
	float milliseconds = 0;

	if (WRITETIME)
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipDeviceSynchronize();
		hipEventRecord(start);
	}

	sort_by_key(keys, keys + current_count, positions);

	if (WRITETIME)
	{
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << "Sorting took " << milliseconds << " milliseconds" << endl;
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
}

int remove_handled(const device_ptr<int> positions, const device_ptr<ullong> keys, const device_ptr<int> destinations,
	const device_ptr<int> helper, const int current_count)
{
	hipEvent_t start;
	hipEvent_t stop;
	float milliseconds = 0;

	if (WRITETIME)
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipDeviceSynchronize();
		hipEventRecord(start);
	}

	const auto iter_start = make_zip_iterator(thrust::make_tuple(keys, positions, destinations));

	const auto iter_end = make_zip_iterator(
		thrust::make_tuple(keys + current_count, positions + current_count, destinations + current_count));

	const auto new_end = remove_if(iter_start, iter_end, helper, equal_to_zero<uchar>());

	if (WRITETIME)
	{
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << "Removing handled took " << milliseconds << " milliseconds" << endl;
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	return new_end - iter_start;
}

int* get_sorted_positions(int* d_positions, const int word_count, unsigned char* d_chars)
{
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds;
	hipEventRecord(start);

	auto keys = device_malloc<ullong>(word_count);
	auto destinations = device_malloc<int>(word_count);
	auto helper = device_malloc<int>(word_count);
	auto output = device_malloc<int>(word_count);

	create_consecutive_numbers(word_count, destinations.get());

	int offset = 0;
	int segment_size = 0;
	int current_count = word_count;

	while (true)
	{
		const int seg_chars = ceil(static_cast<double>(segment_size) / CHARBITS);
		create_hashes_with_seg(keys.get(), d_chars, d_positions, helper.get(), offset, segment_size, current_count, seg_chars);
		offset += CHARSTOHASH - seg_chars;

		sort_wrapper(keys, device_ptr<int>(d_positions), current_count);

		mark_singletons(keys.get(), d_positions, destinations.get(), helper.get(), output.get(), current_count);

		current_count = remove_handled(device_ptr<int>(d_positions), keys, destinations, helper, current_count);
		if (current_count == 0)
			break;

		flags_different_than_last(keys.get(), helper.get(), current_count);
		inclusive_scan(helper, helper + current_count, helper);
		segment_size = compute_segment_size(helper.get(), current_count);
	}

	device_free(keys);
	device_free(destinations);
	device_free(helper);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "All sorting took " << milliseconds << " milliseconds" << endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return output.get();
}
